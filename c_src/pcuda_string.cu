#include <vector>

#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "pcuda_string.h"

PCudaString::PCudaString() {
    this->len = -1;
    this->str = NULL;
}

PCudaString::PCudaString(const std::string& other) {
    this->len = other.length();
    this->ptr = thrust::device_malloc<char>(this->len + 1);
    this->str = raw_pointer_cast(this->ptr);
    hipMemcpy(this->str, other.c_str(), this->len, hipMemcpyHostToDevice);
}

inline PCudaString::PCudaString(const PCudaString& other) {
    this->len = other.len;
    this->str = other.str;
    this->ptr = other.ptr;
}

int PCudaString::length() {
    return this->len;
}

int PCudaString::cstr_length() {
    return this->len + 1;
}

PCudaString::operator std::string() {
    std::string retval;
    thrust::copy(this->ptr, this->ptr + this->len, back_inserter(retval));
    return retval;
}


void PCudaString::destroy() {
    if (this->str) {
        thrust::device_free(this->ptr);
        this->str = NULL;
        this->len = -1;
    }
}

bool operator< (PCudaString lhs, PCudaString rhs) {
    char *l = lhs.str;
    char *r = rhs.str;
    while((*l && *r) && *l == *r) {
        ++l;
        ++r;
    }
    return *l < *r;
}
