#include <vector>
#include <stdio.h>

#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/binary_search.h>
#include <thrust/set_operations.h>
#include <thrust/extrema.h>

#include "pcuda_string.h"

PCudaString::PCudaString() {
    this->len = -1;
    this->str = NULL;
}

PCudaString::PCudaString(const std::string& other) {
    this->len = other.length();
    this->ptr = thrust::device_malloc<char>(this->len + 1);
    this->str = raw_pointer_cast(this->ptr);
    hipMemcpy(this->str, other.c_str(), this->len, hipMemcpyHostToDevice);
}

PCudaString::PCudaString(const PCudaString& other) {
    this->len = other.len;
    this->str = other.str;
    this->ptr = other.ptr;
}

int PCudaString::length() {
    return this->len;
}

int PCudaString::cstr_length() {
    return this->len + 1;
}

PCudaString::operator std::string() {
    std::string retval;
    thrust::copy(this->ptr, this->ptr + this->len, back_inserter(retval));
    return retval;
}


void PCudaString::destroy() {
    if (this->str) {
        thrust::device_free(this->ptr);
        this->str = NULL;
        this->len = -1;
    }
}

bool operator< (PCudaString lhs, PCudaString rhs) {
    char *l = lhs.str;
    char *r = rhs.str;
    while((*l && *r) && *l == *r) {
        ++l;
        ++r;
    }
    return *l < *r;
}

bool pcuda_integer_sort(std::vector<long> *data) {
    thrust::device_vector<long> device = *data;
    thrust::sort(device.begin(), device.end());
    thrust::copy(device.begin(), device.end(), data->begin());
    return true;
}

bool pcuda_float_sort(std::vector<double> *data) {
    thrust::device_vector<double> device = *data;
    thrust::sort(device.begin(), device.end());
    thrust::copy(device.begin(), device.end(), data->begin());
    return true;
}

bool pcuda_string_sort(std::vector<std::string> *data) {
    printf("In pcuda_string_sort\n");
    thrust::device_vector<PCudaString> device;
    printf("Reserving memory\n");
    device.reserve(data->size());
    printf("Copying data to device\n");
    for (std::vector<std::string>::iterator iter = data->begin();
         iter != data->end(); ++iter) {
        std::string s = *iter;
        device.push_back(s);
    }
    printf("On-device sort\n");
    thrust::sort(device.begin(), device.end());
    printf("Copying data from device\n");
    thrust::host_vector<PCudaString> results = device;
    data->clear();
    for (thrust::host_vector<PCudaString>::iterator iter = results.begin();
         iter != results.end(); ++iter) {
        PCudaString cs = *iter;
        std::string s = cs;
        cs.destroy();
        data->push_back(s);
    }
    printf("Done!\n");
    return true;
}

bool pcuda_integer_binary_search(std::vector<long> *data, long target) {
    thrust::device_vector<long> device = *data;
    return thrust::binary_search(device.begin(), device.end(), target, thrust::less<long>());
}

bool pcuda_float_binary_search(std::vector<double> *data, double target) {
    thrust::device_vector<double> device = *data;
    return thrust::binary_search(device.begin(), device.end(), target, thrust::less<double>());
}

void pcuda_integer_intersection(std::vector<long> *first, std::vector<long> *second,
                                std::vector<long> *intersection) {
    thrust::set_intersection(first->begin(), first->end(),
                             second->begin(), second->end(), std::back_inserter(*intersection));
}

void pcuda_float_intersection(std::vector<double> *first, std::vector<double> *second,
                                std::vector<double> *intersection) {
    thrust::set_intersection(first->begin(), first->end(),
                             second->begin(), second->end(), std::back_inserter(*intersection));
}

void pcuda_integer_minmax(std::vector<long> *data, long *minmax) {
    thrust::pair<std::vector<long>::iterator,
                 std::vector<long>::iterator> result = thrust::minmax_element(data->begin(), data->end());
    minmax[0] = *result.first;
    minmax[1] = *result.second;
}

void pcuda_float_minmax(std::vector<double> *data, double *minmax) {
    thrust::pair<std::vector<double>::iterator,
                 std::vector<double>::iterator> result = thrust::minmax_element(data->begin(), data->end());
    minmax[0] = *result.first;
    minmax[1] = *result.second;
}
